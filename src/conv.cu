#include <iostream>
#include <anns_dataset.hpp>
#include <raft/neighbors/cagra_serialize.cuh>
#include <raft/neighbors/cagra.cuh>

template <class data_t, class index_t>
void core(
  const std::string input_path,
  const std::string output_path
  ) {

  raft::device_resources resource_handle;
  auto cagra_index = raft::neighbors::cagra::deserialize<data_t, index_t>(resource_handle, input_path);

  const auto num_graph_elements = cagra_index.graph_degree() * cagra_index.size();
  std::vector<data_t> graph(num_graph_elements);
  hipMemcpy(graph.data(), cagra_index.graph().data_handle(), sizeof(data_t) * num_graph_elements, hipMemcpyDefault);

  mtk::anns_dataset::store(output_path, cagra_index.size(), cagra_index.graph_degree(), graph.data(), mtk::anns_dataset::format_t::FORMAT_BIGANN | mtk::anns_dataset::format_t::HEADER_U32, true);
}

int main(int argc, char** argv) {
  if (argc < 2) {
    std::printf(
      "Extract a CAGRA graph from a RAFT CAGRA index and save it as an internal CAGRA format\n"
      "Usage : %s [path/to/input/raft/cagra/index] [dataset_dtype: float | int8 | uint8] [path/to/output/graph/file.cagra]\n",
      argv[0]
      );
    return 1;
  }

  const std::string input_raft_index_path = argv[1];
  const std::string dtype = argv[2];
  const std::string output_graph_path = argv[3];

  if (dtype == "float") {
    core<float, std::uint32_t>(input_raft_index_path, output_graph_path);
  } else if (dtype == "int8") {
    core<std::int8_t, std::uint32_t>(input_raft_index_path, output_graph_path);
  } else if (dtype == "uint8") {
    core<std::uint8_t, std::uint32_t>(input_raft_index_path, output_graph_path);
  } else {
    std::printf("Unsupported data type : %s\n", dtype.c_str());
  }
}
